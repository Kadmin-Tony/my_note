
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void print(int t)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if( n < t )
        printf("%d\n", n);
}

int main()
{
    // 一个线程块最多有1024个线程
    print<<<3, 4>>>(10);
    hipDeviceSynchronize();

    system("pause");
    return 0;
}