
#include <hip/hip_runtime.h>
#include <stdio.h>
// 使用得默认cuda流可使多个核函数并发执行

__global__ void printNumber(int number)
{
    int n = 0;
    for(int i = 0; i < number+10; i++){
        n += i;
        printf("%d ", n);
    }
    printf("---%d\n", number);
}

int main()
{
  for (int i = 0; i < 20; ++i)
  {
    hipStream_t stream;
    hipStreamCreate(&stream);

    printNumber<<<1, 1, 0, stream>>>(i);
    hipStreamDestroy(stream);
  }
  hipDeviceSynchronize();
}