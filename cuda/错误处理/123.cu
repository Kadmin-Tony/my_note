
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void print(int a)
{
    printf("%d\n", 100 / a);
}

int main()
{
    print<<<1, 1025>>>(4);
    hipError_t err = hipGetLastError();
    printf("%s\n", hipGetErrorString(err));

    err = hipDeviceSynchronize();
    printf("%s\n", hipGetErrorString(err));
    
    return 0;
}