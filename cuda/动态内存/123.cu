
#include <hip/hip_runtime.h>
#include <stdio.h>

// maloc分配的内存cuda无法使用

__global__ void print(int *a, int* b)
{
    a[0] = 1;
    printf("%d", a[0]);
    b[0] = 2;
    printf("%d", b[0]);
}

int main()
{
    int *a, *b;
    size_t size = 100 * sizeof(int);

    hipMallocManaged(&a, size);
    b = (int*) malloc(size);

    print<<<1, 1>>>(a, b);

    hipFree(a);
    free(b);

    hipDeviceSynchronize();
}