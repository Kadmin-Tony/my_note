#include<stdio.h>
// hipMallocManaged()后，cpu或显卡访问内存时，若数据在显卡或cpu上时
// 则会产生页错误，发生内存迁移，占用大量时间
// 若已知即将发生页错误，可使用hipMemPrefetchAsync提前进行 异步 内存迁移

int deviceId;
hipGetDevice(&deviceId);                                         // The ID of the currently active GPU device.

hipMemPrefetchAsync(pointerToSomeUMData, size, deviceId);        // Prefetch to GPU device.
hipMemPrefetchAsync(pointerToSomeUMData, size, hipCpuDeviceId); // Prefetch to host. `hipCpuDeviceId` is a
                                                                  // built-in CUDA variable.

// 将统一内存sum拷贝到主机内存h_sum上
hipMemcpy(h_sum, sum, size * sizeof(int), hipMemcpyDeviceToHost); 