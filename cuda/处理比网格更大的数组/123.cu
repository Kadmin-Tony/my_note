
#include <hip/hip_runtime.h>
#include <stdio.h>

// blockDim.x: 一个线程块有多少个线程
// gridDim.x: 一共有多少个线程块

__global__ void init(int * a, int n)
{
    // 第q个线程
    int q = threadIdx.x + blockDim.x * blockIdx.x; 
    // 总共的线程数 
    int w = gridDim.x * blockDim.x;  
    for(int i = q; i < n; i += w){
        a[i] = i;
    }
}

__global__ void print(int * a, int n)
{
    for(int i = 0; i < 1024; i++){
        printf("%d\n", a[i]);
    }
}

int main()
{
    int *a;
    hipMallocManaged(&a, 1145 * sizeof(int));

    init<<<100, 20>>>(a, 1145);
    print<<<1, 1>>>(a, 1145);

    hipFree(a);
    hipDeviceSynchronize();
}